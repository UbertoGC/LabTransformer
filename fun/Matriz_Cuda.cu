#include "hip/hip_runtime.h"
#include "Matriz2D.h"

#define BLOCK_SIZE 16

// ============================================
// Kernel: Multiplicación de matrices (float)
// ============================================
__global__ void matMulKernel(const float* A, const float* B, float* C,
                             int filasA, int colsA, int colsB) {
    __shared__ float tileA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tileB[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    float val = 0.0f;

    for (int t = 0; t < (colsA + BLOCK_SIZE - 1) / BLOCK_SIZE; ++t) {
        if (row < filasA && (t * BLOCK_SIZE + threadIdx.x) < colsA)
            tileA[threadIdx.y][threadIdx.x] = A[row * colsA + t * BLOCK_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < colsB && (t * BLOCK_SIZE + threadIdx.y) < colsA)
            tileB[threadIdx.y][threadIdx.x] = B[(t * BLOCK_SIZE + threadIdx.y) * colsB + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
            val += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];

        __syncthreads();
    }

    if (row < filasA && col < colsB)
        C[row * colsB + col] = val;
    
        
}

// ============================================
// Kernel: ReLU
// ============================================
__global__ void reluKernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) data[idx] = fmaxf(0.0f, data[idx]);
}

// ============================================
// Kernel: Softmax por filas
// ============================================
__global__ void softmaxKernel(float* A, int rows, int cols) {
    extern __shared__ float shared[];
    int row = blockIdx.x;
    int tid = threadIdx.x;

    if (row >= rows) return;
    float* rowData = A + row * cols;

    // Max
    float maxVal = -1e30f;
    for (int j = tid; j < cols; j += blockDim.x)
        maxVal = fmaxf(maxVal, rowData[j]);
    shared[tid] = maxVal;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) shared[tid] = fmaxf(shared[tid], shared[tid + stride]);
        __syncthreads();
    }
    maxVal = shared[0];

    // Exp y suma
    float sum = 0.0f;
    for (int j = tid; j < cols; j += blockDim.x) {
        rowData[j] = expf(rowData[j] - maxVal);
        sum += rowData[j];
    }
    shared[tid] = sum;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) shared[tid] += shared[tid + stride];
        __syncthreads();
    }
    sum = shared[0];

    for (int j = tid; j < cols; j += blockDim.x)
        rowData[j] /= sum;
}

// ============================================
// Kernel: Normalización por filas
// ============================================
__global__ void normalizeKernel(float* A, int rows, int cols) {
    int row = blockIdx.x;
    if (row >= rows) return;

    extern __shared__ float shared[];
    float* mean = shared;
    float* var = shared + 1;

    if (threadIdx.x == 0) {
        *mean = 0.0f;
        *var = 0.0f;
    }
    __syncthreads();

    atomicAdd(mean, A[row * cols + threadIdx.x]);
    __syncthreads();

    if (threadIdx.x == 0) *mean /= cols;
    __syncthreads();

    float diff = A[row * cols + threadIdx.x] - *mean;
    atomicAdd(var, diff * diff);
    __syncthreads();

    if (threadIdx.x == 0) *var = sqrtf(*var / cols);
    __syncthreads();

    if (*var > 0)
        A[row * cols + threadIdx.x] = diff / *var;
}

// ============================================
// Wrappers CUDA en Matriz2D
// ============================================
void Matriz2D::RELU_CUDA() {
    size_t size = filas * columnas;
    float *d_data;
    CUDA_CHECK(hipMalloc(&d_data, size * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_data, datos, size * sizeof(float), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    reluKernel<<<blocks, threads>>>(d_data, size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(datos, d_data, size * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_data));
}

void Matriz2D::SoftmaxFilas_CUDA() {
    size_t size = filas * columnas;
    float *d_data;
    CUDA_CHECK(hipMalloc(&d_data, size * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_data, datos, size * sizeof(float), hipMemcpyHostToDevice));

    int threads = 256;
    size_t shared_mem = threads * sizeof(float);
    softmaxKernel<<<filas, threads, shared_mem>>>(d_data, filas, columnas);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(datos, d_data, size * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_data));
}

Matriz2D Matriz2D::MultiplicarCUDA(const Matriz2D& B) const {
    if (columnas != B.filas)
        throw std::invalid_argument("Dimensiones incompatibles para multiplicación");

    Matriz2D R(filas, B.columnas);

    float *d_A, *d_B, *d_C;
    size_t sizeA = filas * columnas * sizeof(float);
    size_t sizeB = B.filas * B.columnas * sizeof(float);
    size_t sizeC = filas * B.columnas * sizeof(float);

    CUDA_CHECK(hipMalloc(&d_A, sizeA));
    CUDA_CHECK(hipMalloc(&d_B, sizeB));
    CUDA_CHECK(hipMalloc(&d_C, sizeC));

    CUDA_CHECK(hipMemcpy(d_A, datos, sizeA, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B.datos, sizeB, hipMemcpyHostToDevice));

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((B.columnas + BLOCK_SIZE - 1) / BLOCK_SIZE,
                (filas + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matMulKernel<<<blocks, threads>>>(d_A, d_B, d_C, filas, columnas, B.columnas);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(R.datos, d_C, sizeC, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    return R;
}
__global__ void normalizeKernelGammaBeta(float* A, const float* gamma, const float* beta,
                                          int rows, int cols, float epsilon) {
    int row = blockIdx.x;
    if (row >= rows) return;

    extern __shared__ float shared[];
    float* sum = shared;
    float* sumSq = shared + 1;

    if (threadIdx.x == 0) {
        *sum = 0.0f;
        *sumSq = 0.0f;
    }
    __syncthreads();

    // Calcular suma y suma de cuadrados
    for (int j = threadIdx.x; j < cols; j += blockDim.x) {
        float val = A[row * cols + j];
        atomicAdd(sum, val);
        atomicAdd(sumSq, val * val);
    }
    __syncthreads();

    float mean = *sum / cols;
    float var = (*sumSq / cols) - (mean * mean);
    float invStd = rsqrtf(var + epsilon);

    // Normalización + γ y β
    for (int j = threadIdx.x; j < cols; j += blockDim.x) {
        int idx = row * cols + j;
        float normVal = (A[idx] - mean) * invStd;
        A[idx] = normVal * gamma[j] + beta[j];
    }
}

void Matriz2D::NormalizarFilas_CUDA(const Matriz2D& gamma, const Matriz2D& beta) {
    if (gamma.fil() != 1 || beta.fil() != 1 || gamma.col() != columnas || beta.col() != columnas) {
        throw std::runtime_error("Dimensiones de gamma/beta no compatibles con NormalizarFilas_CUDA");
    }

    int size = filas * columnas;
    float *d_data, *d_gamma, *d_beta;

    CUDA_CHECK(hipMalloc(&d_data, size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_gamma, columnas * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_beta, columnas * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_data, datos, size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_gamma, gamma.Datos(), columnas * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_beta, beta.Datos(), columnas * sizeof(float), hipMemcpyHostToDevice));

    int threads = 256;
    size_t shared_mem = 2 * sizeof(float); // sum y sumSq
    normalizeKernelGammaBeta<<<filas, threads, shared_mem>>>(d_data, d_gamma, d_beta, filas, columnas, 1e-6f);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(datos, d_data, size * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(d_data);
    hipFree(d_gamma);
    hipFree(d_beta);
}


__global__ void KernelSumarFila(float* datos, const float* bias, int filas, int columnas) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < filas * columnas) {
        int col = idx % columnas;
        datos[idx] += bias[col];
    }
}
__global__ void KernelSumarMatrices(float* A, const float* B, int total) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total) {
        A[idx] += B[idx];
    }
}


void Matriz2D::SumarFilaCUDA(const Matriz2D& fila) {
    if (fila.fil() != 1 && fila.fil() != filas) {
        std::cerr << "Error: La matriz no es compatible para broadcast en GPU." << std::endl;
        return;
    }

    int size = filas * columnas;
    int threads = 256;
    int blocks = (size + threads - 1) / threads;

    float* d_datos;
    float* d_bias;
    CUDA_CHECK(hipMalloc(&d_datos, size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_bias, columnas * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_datos, datos, size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_bias, fila.Datos(), columnas * sizeof(float), hipMemcpyHostToDevice));

    KernelSumarFila<<<blocks, threads>>>(d_datos, d_bias, filas, columnas);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(datos, d_datos, size * sizeof(float), hipMemcpyDeviceToHost));
    hipFree(d_datos);
    hipFree(d_bias);
}
void Matriz2D::SumarMatrizCUDA(const Matriz2D& otra) {
    if (filas != otra.Filas() || columnas != otra.Columnas()) {
        throw std::runtime_error("Dimensiones incompatibles en SumarMatrizCUDA");
    }

    int total = filas * columnas;
    float *d_A, *d_B;

    CUDA_CHECK(hipMalloc(&d_A, total * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B, total * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_A, datos, total * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, otra.Datos(), total * sizeof(float), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (total + threads - 1) / threads;

    KernelSumarMatrices<<<blocks, threads>>>(d_A, d_B, total);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(datos, d_A, total * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(d_A);
    hipFree(d_B);
}

__global__ void KernelEscalar(float* datos, float escalar, int total) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total) {
        datos[idx] *= escalar;
    }
}

void Matriz2D::EscalarCUDA(float escalar) {
    int total = filas * columnas;
    float* d_datos;
    hipMalloc(&d_datos, total * sizeof(float));
    hipMemcpy(d_datos, datos, total * sizeof(float), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (total + threads - 1) / threads;
    KernelEscalar<<<blocks, threads>>>(d_datos, escalar, total);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
    hipDeviceSynchronize();

    hipMemcpy(datos, d_datos, total * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_datos);
}
__global__ void DerSoftmaxFilasKernel(const float* softmax_output, const float* grad_sig, 
                                     float* output, int filas, int columnas) {
    int i = blockIdx.x;
    int j = threadIdx.x;
    
    if (i < filas && j < columnas) {
        // Suma reducida (para el término sum(grad_sig * softmax_output))
        __shared__ float sum_shared;
        if (threadIdx.x == 0) {
            sum_shared = 0.0f;
            for (int k = 0; k < columnas; k++) {
                sum_shared += grad_sig[i * columnas + k] * softmax_output[i * columnas + k];
            }
        }
        __syncthreads();

        output[i * columnas + j] = softmax_output[i * columnas + j] * 
                                  (grad_sig[i * columnas + j] - sum_shared);
    }
}

void Matriz2D::DerSoftmaxFilasCUDA(const Matriz2D& grad_sig, Matriz2D& output) {
    float *d_softmax, *d_grad, *d_output;
    CUDA_CHECK(hipMalloc(&d_softmax, filas * columnas * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_grad, filas * columnas * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, filas * columnas * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_softmax, datos, filas * columnas * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_grad, grad_sig.Datos(), filas * columnas * sizeof(float), hipMemcpyHostToDevice));

    dim3 blocks(filas);
    dim3 threads(columnas);
    DerSoftmaxFilasKernel<<<blocks, threads>>>(d_softmax, d_grad, d_output, filas, columnas);

    CUDA_CHECK(hipMemcpy(output.Datos(), d_output, filas * columnas * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_softmax));
    CUDA_CHECK(hipFree(d_grad));
    CUDA_CHECK(hipFree(d_output));
}

__global__ void DerRELUKernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = (input[idx] > 0.0f) ? 1.0f : 0.0f;
    }
}

void Matriz2D::DerRELU_CUDA(const Matriz2D& input, Matriz2D& output) {
    float *d_input, *d_output;
    int size = filas * columnas;
    CUDA_CHECK(hipMalloc(&d_input, size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, size * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_input, input.Datos(), size * sizeof(float), hipMemcpyHostToDevice));

    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    DerRELUKernel<<<numBlocks, blockSize>>>(d_input, d_output, size);

    CUDA_CHECK(hipMemcpy(output.Datos(), d_output, size * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
}
__global__ void CuadradoKernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) data[idx] = data[idx] * data[idx];
}

void Matriz2D::ElementWiseCuadradoCUDA() {
    float* d_data;  // Puntero a memoria en GPU
    int size = filas * columnas;

    // 1. Reservar memoria en GPU
    CUDA_CHECK(hipMalloc(&d_data, size * sizeof(float)));

    // 2. Copiar datos de CPU a GPU
    CUDA_CHECK(hipMemcpy(d_data, datos, size * sizeof(float), hipMemcpyHostToDevice));

    // 3. Configurar y lanzar el kernel
    int blockSize = 256;  // Hilos por bloque (óptimo para la mayoría de GPUs)
    int numBlocks = (size + blockSize - 1) / blockSize;  // Bloques necesarios
    CuadradoKernel<<<numBlocks, blockSize>>>(d_data, size);

    // 4. Copiar resultados de GPU a CPU
    CUDA_CHECK(hipMemcpy(datos, d_data, size * sizeof(float), hipMemcpyDeviceToHost));

    // 5. Liberar memoria de GPU
    CUDA_CHECK(hipFree(d_data));
}

__global__ void RaizKernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) data[idx] = sqrtf(data[idx]);
}
void Matriz2D::ElementWiseRaizCUDA() {
    float* d_data;  // Puntero a memoria en GPU
    int size = filas * columnas;

    // 1. Reservar memoria en GPU
    CUDA_CHECK(hipMalloc(&d_data, size * sizeof(float)));

    // 2. Copiar datos de CPU a GPU
    CUDA_CHECK(hipMemcpy(d_data, datos, size * sizeof(float), hipMemcpyHostToDevice));

    // 3. Configurar y lanzar el kernel
    int blockSize = 256;  // Hilos por bloque (valor óptimo para la mayoría de GPUs)
    int numBlocks = (size + blockSize - 1) / blockSize;  // Bloques necesarios
    RaizKernel<<<numBlocks, blockSize>>>(d_data, size);

    // 4. Copiar resultados de GPU a CPU
    CUDA_CHECK(hipMemcpy(datos, d_data, size * sizeof(float), hipMemcpyDeviceToHost));

    // 5. Liberar memoria de GPU
    CUDA_CHECK(hipFree(d_data));
}